
#include <hip/hip_runtime.h>
/* Furthest point sampling GPU implementation
 * Original author: Haoqiang Fan
 * Modified by Charles R. Qi
 * All Rights Reserved. 2017. 
 */

__global__ void cumsumKernel(int b,int n,const float * __restrict__ inp,float * __restrict__ out){
  const int BlockSize=2048;
  const int paddingLevel=5;
  __shared__ float buffer4[BlockSize*4];
  __shared__ float buffer[BlockSize+(BlockSize>>paddingLevel)];
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    float runningsum=0,runningsum2=0;
    for (int j=0;j<n;j+=BlockSize*4){
      int n24_i=min(n-j,BlockSize*4);
      int n24=(n24_i+3)&~3;
      int n2=n24>>2;
      for (int k=threadIdx.x*4;k<n24_i;k+=blockDim.x*4){
        if (k+3<n24_i){
          float v1=inp[i*n+j+k];
          float v2=inp[i*n+j+k+1];
          v2+=v1;
          float v3=inp[i*n+j+k+2];
          float v4=inp[i*n+j+k+3];
          v4+=v3;
          v3+=v2;
          v4+=v2;
          buffer4[k]=v1;
          buffer4[k+1]=v2;
          buffer4[k+2]=v3;
          buffer4[k+3]=v4;
          buffer[(k>>2)+(k>>(2+paddingLevel))]=v4;
        }else{
          float v=0;
          for (int k2=k;k2<n24_i;k2++){
            v+=inp[i*n+j+k2];
            buffer4[k2]=v;
          }
          for (int k2=n24_i;k2<n24;k2++){
            buffer4[k2]=v;
          }
          buffer[(k>>2)+(k>>(2+paddingLevel))]=v;
        }
      }
      int u=0;
      for (;(2<<u)<=n2;u++){
        __syncthreads();
        for (int k=threadIdx.x;k<int(n2>>(u+1));k+=blockDim.x){
          int i1=(((k<<1)+2)<<u)-1;
          int i2=(((k<<1)+1)<<u)-1;
          i1+=i1>>paddingLevel;
          i2+=i2>>paddingLevel;
          buffer[i1]+=buffer[i2];
        }
      }
      u--;
      for (;u>=0;u--){
        __syncthreads();
        for (int k=threadIdx.x;k<int((n2-(1<<u))>>(u+1));k+=blockDim.x){
          int i1=(((k<<1)+3)<<u)-1;
          int i2=(((k<<1)+2)<<u)-1;
          i1+=i1>>paddingLevel;
          i2+=i2>>paddingLevel;
          buffer[i1]+=buffer[i2];
        }
      }
      __syncthreads();
      for (int k=threadIdx.x*4;k<n24;k+=blockDim.x*4){
        if (k!=0){
          int k2=((k>>2)-1)+(((k>>2)-1)>>paddingLevel);
          buffer4[k]+=buffer[k2];
          buffer4[k+1]+=buffer[k2];
          buffer4[k+2]+=buffer[k2];
          buffer4[k+3]+=buffer[k2];
        }
      }
      __syncthreads();
      for (int k=threadIdx.x;k<n24_i;k+=blockDim.x){
        out[i*n+j+k]=buffer4[k]+runningsum;
      }
      float t=buffer[(n2-1)+((n2-1)>>paddingLevel)]+runningsum2;
      float r2=runningsum+t;
      runningsum2=t-(r2-runningsum);
      runningsum=r2;
      __syncthreads();
    }
  }
}

__global__ void binarysearchKernel(int b,int n,int m,const float * __restrict__ dataset,const float * __restrict__ query, int * __restrict__ result){
  int base=1;
  while (base<n)
    base<<=1;
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    for (int j=blockIdx.y*blockDim.x+threadIdx.x;j<m;j+=blockDim.x*gridDim.y){
      float q=query[i*m+j]*dataset[i*n+n-1];
      int r=n-1;
      for (int k=base;k>=1;k>>=1)
        if (r>=k && dataset[i*n+r-k]>=q)
          r-=k;
      result[i*m+j]=r;
    }
  }
}
#define BufferSize 3072
__global__ void farthestpointsamplingKernel(int b,int n,int c,int m,const float * __restrict__ dataset,float * __restrict__ temp,int * __restrict__ idxs){
  if (m<=0)
    return;
  const int BlockSize=512;
  __shared__ float dists[BlockSize];
  __shared__ int dists_i[BlockSize];
  extern __shared__ float buf[];
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    int old=0;
    if (threadIdx.x==0)
      idxs[i*m+0]=old;
    for (int j=threadIdx.x;j<n;j+=blockDim.x){
      temp[blockIdx.x*n+j]=1e38;
    }
    for (int j=threadIdx.x;j<min(BufferSize,n)*c;j+=blockDim.x){
      buf[j]=dataset[i*n*c+j];
    }
    __syncthreads();
    for (int j=1;j<m;j++){
      int besti=0;
      float best=-1;
      int Ind1=i*n*c+old*c;
      for (int k=threadIdx.x;k<n;k+=blockDim.x){
        float td=temp[blockIdx.x*n+k];
        int Ind2=k<BufferSize?k*c:i*n*c+k*c;
        float d=0;
        for(int iter_c=0;iter_c<c;iter_c++)
            d+=(dataset[Ind2+iter_c]-dataset[Ind1+iter_c])*(dataset[Ind2+iter_c]-dataset[Ind1+iter_c]);///Will abs work the same and be faster?
        float d2=min(d,td);
        if (d2!=td)
          temp[blockIdx.x*n+k]=d2;
        if (d2>best){
          best=d2;
          besti=k;
        }
      }
      dists[threadIdx.x]=best;
      dists_i[threadIdx.x]=besti;
      for (int u=0;(1<<u)<blockDim.x;u++){
        __syncthreads();
        if (threadIdx.x<(blockDim.x>>(u+1))){
          int i1=(threadIdx.x*2)<<u;
          int i2=(threadIdx.x*2+1)<<u;
          if (dists[i1]<dists[i2]){
            dists[i1]=dists[i2];
            dists_i[i1]=dists_i[i2];
          }
        }
      }
      __syncthreads();
      old=dists_i[0];
      if (threadIdx.x==0)
        idxs[i*m+j]=old;
    }
  }
}

__global__ void gatherpointKernel(int b,int n,int c,int m,const float * __restrict__ inp,const int * __restrict__ idx,float * __restrict__ out){
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    for (int j=blockIdx.y*blockDim.x+threadIdx.x;j<m;j+=blockDim.x*gridDim.y){
      int a=idx[i*m+j];
      for(int c_index=0;c_index<c;c_index++)
          out[(i*m+j)*c+c_index]=inp[(i*n+a)*c+c_index];
    }
  }
}

__global__ void scatteraddpointKernel(int b,int n,int m,const float * __restrict__ out_g,const int * __restrict__ idx,float * __restrict__ inp_g){
  for (int i=blockIdx.x;i<b;i+=gridDim.x){
    for (int j=blockIdx.y*blockDim.x+threadIdx.x;j<m;j+=blockDim.x*gridDim.y){
      int a=idx[i*m+j];
      atomicAdd(&inp_g[(i*n+a)*3+0],out_g[(i*m+j)*3+0]);
      atomicAdd(&inp_g[(i*n+a)*3+1],out_g[(i*m+j)*3+1]);
      atomicAdd(&inp_g[(i*n+a)*3+2],out_g[(i*m+j)*3+2]);
    }
  }
}

void cumsumLauncher(int b,int n,const float * inp,float * out){
  cumsumKernel<<<32,512>>>(b,n,inp,out);
}
//require b*n working space
void probsampleLauncher(int b,int n,int m,const float * inp_p,const float * inp_r,float * temp,int * out){
  cumsumKernel<<<32,512>>>(b,n,inp_p,temp);
  binarysearchKernel<<<dim3(32,8,1),512>>>(b,n,m,temp,inp_r,out);
}
//require 32*n working space
void farthestpointsamplingLauncher(int b,int n,int c,int m,const float * inp,float * temp,int * out){
  farthestpointsamplingKernel<<<32,512,BufferSize*c>>>(b,n,c,m,inp,temp,out);
}
void gatherpointLauncher(int b,int n,int c,int m,const float * inp,const int * idx,float * out){
  gatherpointKernel<<<dim3(2,8,1),512>>>(b,n,c,m,inp,idx,out);
}
void scatteraddpointLauncher(int b,int n,int m,const float * out_g,const int * idx,float * inp_g){
  scatteraddpointKernel<<<dim3(2,8,1),512>>>(b,n,m,out_g,idx,inp_g);
}

