
#include <hip/hip_runtime.h>
// input: radius (1), nsample (1), xyz1 (b,n,c), xyz2 (b,m,c)
// output: idx (b,m,nsample), pts_cnt (b,m)
__global__ void knn_gpu(int b, int n, int c, int m, int nsample, const float *xyz1, const float *xyz2, int *idx, float *dist) {
    int batch_index = blockIdx.x;
    xyz1 += n*c*batch_index;
    xyz2 += m*c*batch_index;
    idx += m*nsample*batch_index;
    dist += m*nsample*batch_index;
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int j=index;j<m;j+=stride) {
        for(int nI=0;nI<nsample;nI++)
        {
            dist[j*nsample+nI]=-1;
            idx[j*nsample+nI]=-1;
        }
        for (int k=0;k<n;++k) {
            float d=0;
            for(int ci=0;ci<c;ci++)
                d+=(xyz2[j*c+ci]-xyz1[k*c+ci])*(xyz2[j*c+ci]-xyz1[k*c+ci]);
    	    d=sqrtf(d);
            if(d<dist[j*nsample+nsample-1]||dist[j*nsample+nsample-1]==-1)
            {
                float d_=d;
                int k_=k;
                bool placed=false;
                for(int nI=0;nI<nsample;nI++)
                {
                    if(placed || d_<=dist[j*nsample+nI]||dist[j*nsample+nI]==-1)
                    {
                        float d_tmp=dist[j*nsample+nI];
                        dist[j*nsample+nI]=d_;
                        d_=d_tmp;
                        int k_tmp=idx[j*nsample+nI];
                        idx[j*nsample+nI]=k_;
                        k_=k_tmp;
                        placed=true;
                    }
                }
            }
        }
    }
}

// input: radius (1), nsample (1), xyz1 (b,n,c), xyz2 (b,m,c)
// output: idx (b,m,nsample), pts_cnt (b,m)
__global__ void query_ball_point_gpu(int b, int n, int c, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*c*batch_index;
    xyz2 += m*c*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        int cnt = 0;
        for (int k=0;k<n;++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float d=0;
            for(int ci=0;ci<c;ci++)
                d+=(xyz2[j*c+ci]-xyz1[k*c+ci])*(xyz2[j*c+ci]-xyz1[k*c+ci]);
    	    d=max(sqrtf(d),1e-20f);
            if (d<radius) {
                if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
                    for (int l=0;l<nsample;++l)
                        idx[j*nsample+l] = k;
                }
                idx[j*nsample+cnt] = k;
                cnt+=1;
            }
        }
        pts_cnt[j] = cnt;
    }
}

// input: points (b,n,c), idx (b,m,nsample)
// output: out (b,m,nsample,c)
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
    }
}

// input: grad_out (b,m,nsample,c), idx (b,m,nsample), 
// output: grad_points (b,n,c)
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m*nsample*batch_index;
    grad_out += m*nsample*c*batch_index;
    grad_points += n*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                 atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
            }
        }
    }
}

// input: k (1), distance matrix dist (b,m,n)
// output: idx (b,m,n), dist_out (b,m,n)
// only the top k results within n are useful
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist+=m*n*batch_index;
    outi+=m*n*batch_index;
    out+=m*n*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // copy from dist to dist_out
    for (int j=index;j<m;j+=stride) {
        for (int s=0;s<n;++s) {
            out[j*n+s] = dist[j*n+s];
            outi[j*n+s] = s;
        }
    }

    float *p_dist;
    for (int j=index;j<m;j+=stride) {
        p_dist = out+j*n;
        // selection sort for the first k elements
        for (int s=0;s<k;++s) {
            int min=s; 
            // find the min
            for (int t=s+1;t<n;++t) {
                if (p_dist[t]<p_dist[min]) {
                    min = t;
                }
            }
            // swap min-th and i-th element
            if (min!=s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j*n+min];
                outi[j*n+min] = outi[j*n+s];
                outi[j*n+s] = tmpi;
            }
        }
    }
}

void knnLauncher(int b, int n, int c, int m, int nsample, const float *xyz1, const float *xyz2, int *idx, float *dist) 
{
    knn_gpu<<<b,1024>>>(b,n,c,m,nsample,xyz1,xyz2,idx,dist);
    //cudaDeviceSynchronize();
}
void queryBallPointLauncher(int b, int n, int c, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    query_ball_point_gpu<<<b,1024>>>(b,n,c,m,radius,nsample,xyz1,xyz2,idx,pts_cnt);
    //cudaDeviceSynchronize();
}
void selectionSortLauncher(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    selection_sort_gpu<<<b,1024>>>(b,n,m,k,dist,outi,out); 
    //cudaDeviceSynchronize();
}
void groupPointLauncher(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out){
    group_point_gpu<<<b,1024>>>(b,n,c,m,nsample,points,idx,out);
    //cudaDeviceSynchronize();
}
void groupPointGradLauncher(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points){
    group_point_grad_gpu<<<b,1024>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //group_point_grad_gpu<<<1,1>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //cudaDeviceSynchronize();
}
